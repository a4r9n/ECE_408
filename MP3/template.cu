#include "hip/hip_runtime.h"
/*
 *                        _oo0oo_
 *                       o8888888o
 *                       88" . "88
 *                       (| -_- |)
 *                       0\  =  /0
 *                     ___/`---'\___
 *                   .' \\|     |// '.
 *                  / \\|||  :  |||// \
 *                 / _||||| -:- |||||- \
 *                |   | \\\  - /// |   |
 *                | \_|  ''\---/''  |_/ |
 *                \  .-\__  '-'  ___/-. /
 *              ___'. .'  /--.--\  `. .'___
 *           ."" '<  `.___\_<|>_/___.' >' "".
 *          | | :  `- \`.;`\ _ /`;.`/ - ` : | |
 *          \  \ `_.   \_ __\ /__ _/   .-` /  /
 *      =====`-.____`.___ \_____/___.-`___.-'=====
 *                        `=---='
 * 
 * 
 *      ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
 * 
 *            佛祖保佑     永不宕机     永无BUG
 */


#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  int TILE_WIDTH = 16;
  __shared__ float subTileM[16][16];
  __shared__ float subTileN[16][16];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Col =  by * TILE_WIDTH +ty;
  int Row =  bx * TILE_WIDTH +tx;
  float Pvalue = 0;
  //int Width = max(numAColumns,numBRows);
  int Width = numAColumns;
  for (int q = 0; q < numAColumns/TILE_WIDTH; ++q) {
// Collaborative loading of M and N tiles into shared memory

    if(q*TILE_WIDTH+tx < numAColumns && Row < numARows ){
      subTileM[ty][tx]=A[Row*Width + q*TILE_WIDTH+tx];
    } else {
      subTileM[ty][tx]=0;
    }

    if(Col < numBColumns && q*TILE_WIDTH+ty < numBRows ){
      subTileN[ty][tx]=B[(q*TILE_WIDTH+ty)*numBColumns+Col];
    } else {
      subTileN[ty][tx]=0;
    }

    __syncthreads();
    if (Row < numCRows && Col < numCColumns) {
        for (int k = 0; k < TILE_WIDTH; ++k) {
        Pvalue += subTileM[ty][k] * subTileN[k][tx];
        __syncthreads();
      }
    }
  }
  if (Row < numCRows && Col < numCColumns) {
    C[Row * numCColumns + Col] = Pvalue;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  wbTime_stop(Generic, "Importing data and creating memory on host");
    hostC = (float *)malloc(numCRows*numCColumns*sizeof(float));

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here

  hipMalloc((void**) &deviceA,numARows*numAColumns*sizeof(float));
  hipMalloc((void**) &deviceB,numBRows*numBColumns*sizeof(float));
  hipMalloc((void**) &deviceC,numCRows*numCColumns*sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows*numAColumns*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows*numBColumns*sizeof(float), hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil((float)numCColumns/16),ceil((float)numCRows/16),1);
  dim3 DimBlock(16,16,1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<DimGrid,DimBlock>>>(deviceA,deviceB,deviceC,
                                        numARows,numAColumns,
                                        numBRows,numBColumns,
                                        numCRows,numCColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC,deviceC,numCRows*numCColumns*sizeof(float),hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
